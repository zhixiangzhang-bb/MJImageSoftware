#include "Cuda.h"
#include "CUDARandom.cuh"
#include <iostream>



int main()
{
    int size = 1024;
    int bytes = size * sizeof(int);

    // 定义并分配主机内存
    int* h_a = new int[size];
    int* h_b = new int[size];
    int* h_c = new int[size];

    // 初始化数组
    for (int i = 0; i < size; i++) {
        h_a[i] = i;
        h_b[i] = i;
    }

    // 定义并分配设备内存
    int* d_a, * d_b, * d_c;
    hipMalloc((void**)&d_a, bytes);
    hipMalloc((void**)&d_b, bytes);
    hipMalloc((void**)&d_c, bytes);

    // 将数据从主机内存复制到设备内存
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    // 定义网格和块的大小
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;

    // 调用CUDA核函数
    addArrays << <gridSize, blockSize >> > (d_a, d_b, d_c, size);

    // 将结果从设备内存复制到主机内存
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // 打印结果
    for (int i = 0; i < size; i++) {
        std::cout << h_a[i] << " + " << h_b[i] << " = " << h_c[i] << std::endl;
    }

    // 释放内存
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    cudnntrain();
    return 0;
}


