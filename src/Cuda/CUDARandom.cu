#include "hip/hip_runtime.h"
#include "CUDARandom.cuh"





// CUDA核函数：将两个数组相加
__global__ void addArrays(int* a, int* b, int* c, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}


